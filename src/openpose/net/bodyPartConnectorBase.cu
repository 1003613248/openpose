#include "hip/hip_runtime.h"
﻿#include <openpose/gpu/cuda.hpp>
#include <openpose/pose/poseParameters.hpp>
#include <openpose/utilities/fastMath.hpp>
#include <openpose/net/bodyPartConnectorBase.hpp>
#include <caffe/caffe.hpp>
#include <openpose/pose/poseParameters.hpp>

namespace op
{
    const dim3 THREADS_PER_BLOCK{4, 16, 16};

    template<typename T>
    inline __device__ int intRoundGPU(const T a)
    {
        return int(a+T(0.5));
    }

    template <typename T>
    inline __device__  T process(const T* bodyPartA, const T* bodyPartB, const T* mapX, const T* mapY,
                                 const int heatmapWidth, const int heatmapHeight, const T interThreshold,
                                 const T interMinAboveThreshold)
    {
        const auto vectorAToBX = bodyPartB[0] - bodyPartA[0];
        const auto vectorAToBY = bodyPartB[1] - bodyPartA[1];
        const auto vectorAToBMax = max(abs(vectorAToBX), abs(vectorAToBY));
        const auto numberPointsInLine = max(5, min(25, intRoundGPU(sqrt(5*vectorAToBMax))));
        const auto vectorNorm = T(sqrt(vectorAToBX*vectorAToBX + vectorAToBY*vectorAToBY));

        if (vectorNorm > 1e-6)
        {
            const auto sX = bodyPartA[0];
            const auto sY = bodyPartA[1];
            const auto vectorAToBNormX = vectorAToBX/vectorNorm;
            const auto vectorAToBNormY = vectorAToBY/vectorNorm;

            auto sum = T(0.);
            auto count = 0;
            const auto vectorAToBXInLine = vectorAToBX/numberPointsInLine;
            const auto vectorAToBYInLine = vectorAToBY/numberPointsInLine;
            for (auto lm = 0; lm < numberPointsInLine; lm++)
            {
                const auto mX = min(heatmapWidth-1, intRoundGPU(sX + lm*vectorAToBXInLine));
                const auto mY = min(heatmapHeight-1, intRoundGPU(sY + lm*vectorAToBYInLine));
                const auto idx = mY * heatmapWidth + mX;
                const auto score = (vectorAToBNormX*mapX[idx] + vectorAToBNormY*mapY[idx]);
                if (score > interThreshold)
                {
                    sum += score;
                    count++;
                }
            }

            // Return PAF score
            if (count/T(numberPointsInLine) > interMinAboveThreshold){
                return sum/count;
            }else
            {
                // Ideally, if distanceAB = 0, PAF is 0 between A and B, provoking a false negative
                // To fix it, we consider PAF-connected keypoints very close to have a minimum PAF score, such that:
                //     1. It will consider very close keypoints (where the PAF is 0)
                //     2. But it will not automatically connect them (case PAF score = 1), or real PAF might got
                //        missing
                const auto l2Dist = sqrtf(vectorAToBX*vectorAToBX + vectorAToBY*vectorAToBY);
                const auto threshold = sqrtf(heatmapWidth*heatmapHeight)/150; // 3.3 for 368x656, 6.6 for 2x resolution
                if (l2Dist < threshold)
                    return T(0.15);
            }
        }
        return -1;
    }

    template <typename T>
    __global__ void pafScoreKernel(T* pairScoresPtr, const T* const heatMapPtr, const T* const peaksPtr,
                                   const unsigned int* const bodyPartPairsPtr, const unsigned int* const mapIdxPtr,
                                   const unsigned int maxPeaks, const int numberBodyPartPairs,
                                   const int heatmapWidth, const int heatmapHeight, const T interThreshold,
                                   const T interMinAboveThreshold)
    {
        const auto pairIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto peakA = (blockIdx.y * blockDim.y) + threadIdx.y;
        const auto peakB = (blockIdx.z * blockDim.z) + threadIdx.z;

        if (pairIndex < numberBodyPartPairs && peakA < maxPeaks && peakB < maxPeaks)
        {
            const auto baseIndex = 2*pairIndex;
            const auto partA = bodyPartPairsPtr[baseIndex];
            const auto partB = bodyPartPairsPtr[baseIndex + 1];

            const T numberPeaksA = peaksPtr[3*partA*(maxPeaks+1)];
            const T numberPeaksB = peaksPtr[3*partB*(maxPeaks+1)];

            const auto outputIndex = (pairIndex*maxPeaks+peakA)*maxPeaks + peakB;
            if (peakA < numberPeaksA && peakB < numberPeaksB)
            {
                const auto mapIdxX = mapIdxPtr[baseIndex];
                const auto mapIdxY = mapIdxPtr[baseIndex + 1];

                const T* const bodyPartA = peaksPtr + (3*(partA*(maxPeaks+1) + peakA+1));
                const T* const bodyPartB = peaksPtr + (3*(partB*(maxPeaks+1) + peakB+1));
                const T* const mapX = heatMapPtr + mapIdxX*heatmapWidth*heatmapHeight;
                const T* const mapY = heatMapPtr + mapIdxY*heatmapWidth*heatmapHeight;
                pairScoresPtr[outputIndex] = process(
                    bodyPartA, bodyPartB, mapX, mapY, heatmapWidth, heatmapHeight, interThreshold,
                    interMinAboveThreshold);
            }
            else
                pairScoresPtr[outputIndex] = -1;
        }
    }

    template <typename T>
    __global__ void tafScoreKernel2(T* tafScoresPtr)
    {

    }

    template <typename T>
    __global__ void tafScoreKernel(T* tafScoresPtr, const T* const heatMapPtr, const T* const posePtr,
                                   const T* const trackletPtr, const int* const tafPartPairsPtr,
                                   const int totalPose, const int totalTracklet,
                                   const int numberBodyParts, const int tafHeatmapOffset,
                                   const int heatmapWidth, const int heatmapHeight, const T interThreshold,
                                   const T interMinAboveThreshold)
    {
        const auto pairIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto pid = (blockIdx.y * blockDim.y) + threadIdx.y;
        const auto tid = (blockIdx.z * blockDim.z) + threadIdx.z;

        if(pid >= totalPose || tid >= totalTracklet) return;

        const auto partA = tafPartPairsPtr[pairIndex*2 + 0];
        const auto partB = tafPartPairsPtr[pairIndex*2 + 1];

        const T* tafMapPtr = heatMapPtr + tafHeatmapOffset*(heatmapWidth*heatmapHeight);
        const T* mapX = tafMapPtr + (2*pairIndex + 0)*(heatmapWidth*heatmapHeight);
        const T* mapY = tafMapPtr + (2*pairIndex + 1)*(heatmapWidth*heatmapHeight);

        const T* bodyPartA = posePtr + (pid * numberBodyParts * 3) + (partA * 3);
        const T* bodyPartB = trackletPtr + (tid * numberBodyParts * 3) + (partB * 3);

        const auto outputIndex = (pairIndex*totalPose*totalTracklet) + (pid*totalTracklet) + tid;

        if(bodyPartA[2] < 0.05 || bodyPartB[2] < 0.05){
            tafScoresPtr[outputIndex] = -1;
        }else{
            tafScoresPtr[outputIndex] = process(
                bodyPartB, bodyPartA, mapX, mapY, heatmapWidth, heatmapHeight, interThreshold,
                interMinAboveThreshold);

//            if(pairIndex == 2){
//                if(tafScoresPtr[outputIndex] != -1){
//                    printf("%d-%d [%f, %f, %f] [%f, %f, %f]    \n", pid, tid, bodyPartA[0], bodyPartA[1], bodyPartA[2], bodyPartB[0], bodyPartB[1], bodyPartB[2]);
//                    printf("%d-%d %f\n", pid, tid, tafScoresPtr[outputIndex]);
//                }
//            }
        }

    }


    template <typename T>
    void tafScoreGPU(const op::Array<T>& poseKeypoints, const op::Array<T>& trackletKeypoints,
                     const std::shared_ptr<caffe::Blob<T>> heatMapsBlob, op::Array<T>& tafScores,
                     const std::vector<int> tafPartPairs, int* &tafPartPairsGpuPtr, int tafChannelStart)
    {
        try
        {
            //static int done = 0;

//            std::cout << poseKeypoints << std::endl;
//            std::cout << trackletKeypoints << std::endl;

            // Tree
            //if(done == 0){
            if(tafPartPairsGpuPtr == nullptr){
                hipMalloc((void **)&tafPartPairsGpuPtr, tafPartPairs.size() * sizeof(int));
                hipMemcpy(tafPartPairsGpuPtr, &tafPartPairs[0], tafPartPairs.size() * sizeof(int),
                           hipMemcpyHostToDevice);
            }

            // Copy both to GPU
            T* poseGpuPtr;
            T* trackletGpuPtr;
            hipMalloc((void **)&poseGpuPtr, poseKeypoints.getVolume() * sizeof(T));
            hipMemcpy(poseGpuPtr, poseKeypoints.getConstPtr(), poseKeypoints.getVolume() * sizeof(T),
                       hipMemcpyHostToDevice);
            hipMalloc((void **)&trackletGpuPtr, trackletKeypoints.getVolume() * sizeof(T));
            hipMemcpy(trackletGpuPtr, trackletKeypoints.getConstPtr(), trackletKeypoints.getVolume() * sizeof(T),
                       hipMemcpyHostToDevice);

            // Score Data
            int totalPairs = (tafPartPairs.size()/2);
            int totalPosePeople = poseKeypoints.getSize(0);
            int totalTrackletPeople = trackletKeypoints.getSize(0);
            int totalComputations = totalPairs * totalPosePeople * totalTrackletPeople;
            T* tafScoreGpuPtr;
            hipMalloc((void **)&tafScoreGpuPtr, totalComputations * sizeof(T));

//            std::cout << totalPairs << std::endl;
//            std::cout << heatMapsBlob->shape_string() << std::endl;

            // Kernel
            const T* heatMapPtr = (T*)heatMapsBlob->gpu_data();
            //const T* heatMapPtr = nullptr;
            int totalBodyParts = poseKeypoints.getSize(1);
            T interThreshold = 0.05;
            T interMinAboveThreshold = 0.95;
            const dim3 numBlocks{
                op::getNumberCudaBlocks(totalPairs, THREADS_PER_BLOCK.x),
                op::getNumberCudaBlocks(op::POSE_MAX_PEOPLE, THREADS_PER_BLOCK.y),
                op::getNumberCudaBlocks(op::TRACK_MAX_PEOPLE, THREADS_PER_BLOCK.z)};

            tafScoreKernel<<<numBlocks, THREADS_PER_BLOCK>>>(
                tafScoreGpuPtr, heatMapPtr, poseGpuPtr, trackletGpuPtr, tafPartPairsGpuPtr,
                totalPosePeople, totalTrackletPeople,
                totalBodyParts, tafChannelStart, (int)heatMapsBlob->shape(3), (int)heatMapsBlob->shape(2), interThreshold, interMinAboveThreshold);


            tafScores.reset({totalPairs, totalPosePeople, totalTrackletPeople});

            hipMemcpy(tafScores.getPtr(), tafScoreGpuPtr, totalComputations * sizeof(T),
                       hipMemcpyDeviceToHost);

//            for(int k=0; k<totalTrackletPeople; k++){
//                for(int j=0; j<totalPosePeople; j++){
//                    std::cout << std::to_string(j) + "-" + std::to_string(k) + "---" + std::to_string(tafScores.at({0,j,k})) << std::endl;
//                }
//            }

//            for(int i=0; i<totalPairs; i++){
//                for(int j=0; j<totalPosePeople; j++){
//                    for(int k=0; k<totalTrackletPeople; k++){
//                        std::cout << std::to_string(j) + "-" + std::to_string(k) + "---" + std::to_string(tafScores.at({i,j,k})) << std::endl;
//                    }
//                }
//            }

//            for(int i=0; i<totalComputations; i++){
//                std::cout << tafScores.getPtr()[i] << std::endl;
//            }

            hipFree(tafScoreGpuPtr);
            hipFree(poseGpuPtr);
            hipFree(trackletGpuPtr);

            // Sanity check
            cudaCheck(__LINE__, __FUNCTION__, __FILE__);
        }
        catch (const std::exception& e)
        {
            error(e.what(), __LINE__, __FUNCTION__, __FILE__);
        }
    }

    template <typename T>
    void connectBodyPartsGpu(Array<T>& poseKeypoints, Array<T>& poseScores, const T* const heatMapGpuPtr,
                             const T* const peaksPtr, const PoseModel poseModel, const Point<int>& heatMapSize,
                             const int maxPeaks, const T interMinAboveThreshold, const T interThreshold,
                             const int minSubsetCnt, const T minSubsetScore, const T scaleFactor,
                             const bool maximizePositives, Array<T> pairScoresCpu, T* pairScoresGpuPtr,
                             const unsigned int* const bodyPartPairsGpuPtr, const unsigned int* const mapIdxGpuPtr,
                             const T* const peaksGpuPtr)
    {
        try
        {
            // Parts Connection
            const auto& bodyPartPairs = getPosePartPairs(poseModel);
            const auto numberBodyParts = getPoseNumberBodyParts(poseModel);
            const auto numberBodyPartPairs = (unsigned int)(bodyPartPairs.size() / 2);
            const auto totalComputations = pairScoresCpu.getVolume();

            if (numberBodyParts == 0)
                error("Invalid value of numberBodyParts, it must be positive, not " + std::to_string(numberBodyParts),
                      __LINE__, __FUNCTION__, __FILE__);
            if (bodyPartPairsGpuPtr == nullptr || mapIdxGpuPtr == nullptr)
                error("The pointers bodyPartPairsGpuPtr and mapIdxGpuPtr cannot be nullptr.",
                      __LINE__, __FUNCTION__, __FILE__);


            // Run Kernel - pairScoresGpu
            const dim3 numBlocks{
                getNumberCudaBlocks(numberBodyPartPairs, THREADS_PER_BLOCK.x),
                getNumberCudaBlocks(maxPeaks, THREADS_PER_BLOCK.y),
                getNumberCudaBlocks(maxPeaks, THREADS_PER_BLOCK.z)};
            pafScoreKernel<<<numBlocks, THREADS_PER_BLOCK>>>(
                pairScoresGpuPtr, heatMapGpuPtr, peaksGpuPtr, bodyPartPairsGpuPtr, mapIdxGpuPtr,
                maxPeaks, (int)numberBodyPartPairs, heatMapSize.x, heatMapSize.y, interThreshold,
                interMinAboveThreshold);
            // pairScoresCpu <-- pairScoresGpu
            hipMemcpy(pairScoresCpu.getPtr(), pairScoresGpuPtr, totalComputations * sizeof(T),
                       hipMemcpyDeviceToHost);

            // New code
            // Get pair connections and their scores
            const auto pairConnections = pafPtrIntoVector(
                pairScoresCpu, peaksPtr, maxPeaks, bodyPartPairs, numberBodyPartPairs);
            const auto peopleVector = pafVectorIntoPeopleVector(
                pairConnections, peaksPtr, maxPeaks, bodyPartPairs, numberBodyParts);

            // // Old code
            // // Get pair connections and their scores
            // // std::vector<std::pair<std::vector<int>, double>> refers to:
            // //     - std::vector<int>: [body parts locations, #body parts found]
            // //     - double: person subset score
            // const T* const tNullptr = nullptr;
            // const auto peopleVector = createPeopleVector(
            //     tNullptr, peaksPtr, poseModel, heatMapSize, maxPeaks, interThreshold, interMinAboveThreshold,
            //     bodyPartPairs, numberBodyParts, numberBodyPartPairs, pairScoresCpu);

            // Delete people below the following thresholds:
                // a) minSubsetCnt: removed if less than minSubsetCnt body parts
                // b) minSubsetScore: removed if global score smaller than this
                // c) maxPeaks (POSE_MAX_PEOPLE): keep first maxPeaks people above thresholds
            int numberPeople;
            std::vector<int> validSubsetIndexes;
            validSubsetIndexes.reserve(fastMin((size_t)maxPeaks, peopleVector.size()));
            removePeopleBelowThresholds(validSubsetIndexes, numberPeople, peopleVector, numberBodyParts, minSubsetCnt,
                                        minSubsetScore, maxPeaks, maximizePositives);

            // Fill and return poseKeypoints
            peopleVectorToPeopleArray(poseKeypoints, poseScores, scaleFactor, peopleVector, validSubsetIndexes,
                                      peaksPtr, numberPeople, numberBodyParts, numberBodyPartPairs);

            // Sanity check
            cudaCheck(__LINE__, __FUNCTION__, __FILE__);
        }
        catch (const std::exception& e)
        {
            error(e.what(), __LINE__, __FUNCTION__, __FILE__);
        }
    }

    template void connectBodyPartsGpu(
        Array<float>& poseKeypoints, Array<float>& poseScores, const float* const heatMapGpuPtr,
        const float* const peaksPtr, const PoseModel poseModel, const Point<int>& heatMapSize, const int maxPeaks,
        const float interMinAboveThreshold, const float interThreshold, const int minSubsetCnt,
        const float minSubsetScore, const float scaleFactor, const bool maximizePositives,
        Array<float> pairScoresCpu, float* pairScoresGpuPtr, const unsigned int* const bodyPartPairsGpuPtr,
        const unsigned int* const mapIdxGpuPtr, const float* const peaksGpuPtr);
    template void connectBodyPartsGpu(
        Array<double>& poseKeypoints, Array<double>& poseScores, const double* const heatMapGpuPtr,
        const double* const peaksPtr, const PoseModel poseModel, const Point<int>& heatMapSize, const int maxPeaks,
        const double interMinAboveThreshold, const double interThreshold, const int minSubsetCnt,
        const double minSubsetScore, const double scaleFactor, const bool maximizePositives,
        Array<double> pairScoresCpu, double* pairScoresGpuPtr, const unsigned int* const bodyPartPairsGpuPtr,
        const unsigned int* const mapIdxGpuPtr, const double* const peaksGpuPtr);

    template void tafScoreGPU(const op::Array<float>& poseKeypoints, const op::Array<float>& trackletKeypoints,
    const std::shared_ptr<caffe::Blob<float>> heatMapsBlob, op::Array<float>& tafScores,
    const std::vector<int> tafPartPairs, int* &tafPartPairsGpuPtr, int tafChannelStart);
    template void tafScoreGPU(const op::Array<double>& poseKeypoints, const op::Array<double>& trackletKeypoints,
    const std::shared_ptr<caffe::Blob<double>> heatMapsBlob, op::Array<double>& tafScores,
    const std::vector<int> tafPartPairs, int* &tafPartPairsGpuPtr, int tafChannelStart);
}
